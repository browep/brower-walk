#include "hip/hip_runtime.h"
#include "brower_walk_cuda.h"
#include <stdio.h>
#include <inttypes.h>

#define BLOCKS_NUM        1
#define THREADS_PER_BLOCK 1

__global__
void brower_walk_init(uint64_cu walk_size, uint64_cu s0, uint64_cu s1, uint64_cu *walk_path)
{
    for (int i = 0; i < walk_size; i++) {
        uint64_cu x = s0;
        uint64_cu const y = s1;
        s0 = y;
        x ^= x << 23; // a
        s1 = x ^ y ^ (x >> 18) ^ (y >> 5); // b, c
        walk_path[i] = s1 + y;
    }
}

__global__
void brower_walk(uint64_cu num_steps, uint64_cu walk_size, uint64_cu * walk_path, uint64_cu * consumed_steps)
{
    uint64_cu next_step = walk_size - 1;

    for (int i = 0; i < num_steps; i++) {
        uint64_t val = walk_path[next_step];
        uint64_t new_val = (val << 1) + (i % 2);
        walk_path[next_step] = new_val;

        consumed_steps[i] = new_val;

        next_step = val % walk_size;
    }
}

static void cuda_errors_handle(void)
{
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
    {
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
        exit(0);
    }
    if (errAsync != hipSuccess)
    {
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
        exit(0);
    }
}

void brower_walk_init_wrapper(uint64_cu walk_size,
                              uint64_cu s0,
                              uint64_cu s1,
                              uint64_cu ** walk_path_gpu)
{
    hipMalloc(walk_path_gpu, walk_size*sizeof(uint64_cu)); 

    brower_walk_init<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(walk_size, s0, s1, *walk_path_gpu);

    // Wait for GPU to finish.
    hipDeviceSynchronize();
    cuda_errors_handle();
}

void brower_walk_wrapper(uint64_cu   num_steps,
                         uint64_cu   walk_size,
                         uint64_cu * walk_path_gpu,
                         uint64_cu * consumed_steps)
{
    uint64_cu * consumed_steps_gpu;
    hipMalloc(&consumed_steps_gpu, num_steps*sizeof(uint64_cu)); 

    brower_walk<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(num_steps, walk_size, walk_path_gpu, consumed_steps_gpu);

    hipMemcpy(consumed_steps, consumed_steps_gpu, num_steps*sizeof(uint64_cu), hipMemcpyDeviceToHost);

    hipFree(walk_path_gpu);
    hipFree(consumed_steps_gpu);
    cuda_errors_handle();
}

